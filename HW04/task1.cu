#include <cstdio>
#include <hip/hip_runtime.h>
#include <random>

#include "matmul.cuh"


int main(int argc, char* argv[]) {

    if(argc != 3){
        exit(-1);
    }

    long unsigned int n = atoi(argv[1]);
    long unsigned int threads_per_block = atoi(argv[2]);
    long unsigned int SIZE = n*n;

    //allocating host memory for input matrices
    float *h_A, *h_B;
    h_A = new float[SIZE];
    h_B = new float[SIZE];

    //random number generation
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dist(-1, 1);

    for(long unsigned int i = 0; i < SIZE; i++){
        h_A[i] = dist(gen);
        h_B[i] = dist(gen);
    }

    // Allocate vectors in device memory
    float *A, *B, *C;
    hipMalloc(&A, SIZE*sizeof(float));
    hipMalloc(&B, SIZE*sizeof(float));
    hipMalloc(&C, SIZE*sizeof(float));

    //copy from host memory to device memory
    hipMemcpy(A, h_A, SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B, h_B, SIZE*sizeof(float), hipMemcpyHostToDevice);

    //creating cuda timing variables
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    //kernel call
    matmul(A, B, C, n, threads_per_block);
    hipEventRecord(stop);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    //allocate host memory for product matrix
    float *h_C;
    h_C = new float[SIZE];

    //copying output from device to host memory
    hipMemcpy(h_C, C, SIZE*sizeof(float), hipMemcpyDeviceToHost);

    //printing output
    
    std::printf("%f\n%f\n", h_C[SIZE - 1], ms);

    hipFree(A);
    hipFree(B);
    hipFree(C);

    delete [] h_A;
    delete [] h_B;
    delete [] h_C;

    return 0;
}