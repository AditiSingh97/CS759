#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>

#include "stencil.cuh"

__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R){
    extern __shared__ float shMem[];
    unsigned int M = 2*R + 1;
    if(threadIdx.x < M){
        shMem[threadIdx.x] = mask[threadIdx.x];
    }
    //blockDimx.x + 2 * R
    if((blockIdx.x  * blockDim.x + threadIdx.x) < n){
        unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
        if(threadIdx.x == 0){
            for(long int i = M+R; i >= M; i--){
                if((threadid - (M + R - i)) < 0){
                    shMem[i] = 1.0;
                }else{
                    shMem[i] = image[threadid - (M + R - i)];
                }
            }
        }else{
            //M (for mask) ; R (left padding) ; blockDim.x (==threads_per_block) (for each output) ; R (right padding)
            if((threadIdx.x == blockDim.x - 1) || (threadid == n-1)){
                if(threadid != n-1)
                {
                    unsigned long offset = M + blockDim.x + R - 1;
                    for(long int i = offset; i <= offset + R; i++){
                        if(threadid + (i - offset) > (n-1)){
                            shMem[i] = 1.0;
                        }else{
                            shMem[i] = image[threadid + (i - offset)];
                        }
                    }
                }else{
                    shMem[M + R + threadIdx.x] = image[threadid];
                    for(long int i = 1; i <= R; i++){
                        shMem[M + threadIdx.x + R + i] = 1.0;
                    }
                }
            }else{
                shMem[M + R + threadIdx.x] = image[threadid];
            }
        }
    }

    __syncthreads();
    unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadid < n){
        int op_off = M + blockDim.x + 2*R + threadIdx.x;
        int pos = M + R + threadIdx.x;
        shMem[op_off] = 0.0;
        signed int neg = (-1) * (signed)R;
        for(signed int j = neg; j <= (signed)R; j++){
            shMem[op_off] += shMem[((signed)pos+j)] * shMem[(j+(signed)R)];
        }
        output[threadid] = shMem[op_off];
    }
}

__host__ void stencil(const float* image, const float* mask, float* output, unsigned int n, unsigned int R, unsigned int threads_per_block){
    //computing the correct number of blocks
    double q = ((double)(n))/((double)threads_per_block);
    long unsigned int num_blocks = ceil(q);
    //kernel call
    stencil_kernel<<<num_blocks, threads_per_block, (4*R+1 + 2*threads_per_block) * sizeof(float)>>>(image, mask, output, n, R);
    hipDeviceSynchronize();
}
