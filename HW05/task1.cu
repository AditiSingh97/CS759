#include "hip/hip_runtime.h"
#include <cstdio>
#include <random>
#include <hip/hip_runtime.h>

#include "reduce.cuh"

int main(int argc, char* argv[]) {
    if(argc != 3){
        exit(-1);
    }

    unsigned int N = atoi(argv[1]);
    unsigned int threads_per_block = atoi(argv[2]);

    //allocating host memory for input matrices
    unsigned int num_blocks = ceil((double)N/(double)(2.0*threads_per_block));
    float *h_A, *h_B;
    h_A = new float[N];
    h_B = new float[num_blocks];

    //random number generation
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dist(-1, 1);

    for(long unsigned int i = 0; i < N; i++){
        h_A[i] = (float)i;
    }

    // Allocate vectors in device memory
    float *A, *B;
    hipMalloc(&A, N*sizeof(float));
    hipMalloc(&B, num_blocks*sizeof(float));

    //copy from host memory to device memory
    hipMemcpy(A, h_A, N*sizeof(float), hipMemcpyHostToDevice);

    //creating cuda timing variables
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    //kernel call
    reduce(&A, &B, N, threads_per_block);
    hipEventRecord(stop);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    hipMemcpy(h_A, A, 1*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_B, B, 1*sizeof(float), hipMemcpyDeviceToHost);
    std::printf("final answer A %f\n", h_A[0]);

    return 0;
}
