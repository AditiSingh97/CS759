#include "hip/hip_runtime.h"
#include <cstdio>
#include <random>
#include <hip/hip_runtime.h>

#include "reduce.cuh"

void verifier(float *A, float *result, unsigned int n){
	double threshold = 1e-5;
	float ground_truth = 0.0;
	for(unsigned int i = 0; i < n; i++){
		ground_truth += A[i];
	}
//	printf("ground_truth: %f, result: %f\n", ground_truth, result[0]);
//	printf("difference between result and ground_truth: %f\n", result[0] - ground_truth);
}

int main(int argc, char* argv[]) {
    if(argc != 3){
        exit(-1);
    }

    unsigned int N = atoi(argv[1]);
    unsigned int threads_per_block = atoi(argv[2]);

    //allocating host memory for input matrices
    unsigned int num_blocks = ceil((double)N/(double)(2.0*threads_per_block));
    float *h_A, *h_B;
    h_A = new float[N];
    h_B = new float[num_blocks];

    //random number generation
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dist(-1, 1);

    for(long unsigned int i = 0; i < N; i++){
        h_A[i] = dist(gen);
    }
    for(unsigned int i = 0; i < num_blocks; i++){
	   h_B[i] = (float)0.0;
    } 

    // Allocate vectors in device memory
    float *A, *B;
    hipMalloc(&A, N*sizeof(float));
    hipMalloc(&B, num_blocks*sizeof(float));

    //copy from host memory to device memory
    hipMemcpy(A, h_A, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B, h_B, num_blocks*sizeof(float), hipMemcpyHostToDevice);

    //creating cuda timing variables
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    //kernel call
    reduce(&A, &B, N, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Get the elapsed time in milliseconds
    float ms3;
    hipEventElapsedTime(&ms3, start, stop);
    printf("%f\n%f\n%f\n", C_double[0], C_double[SIZE-1], ms3);

    hipFree(A_double);
    hipFree(B_double);
    hipFree(C_double);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    float *result = new float;
    hipMemcpy(result, A, 1*sizeof(float), hipMemcpyDeviceToHost);
    printf("%.6f\n%.6f\n", result[0], ms);
    hipFree(A);
    hipFree(B);
    delete [] h_A;
    delete [] h_B;
    delete result;
    return 0;
}
