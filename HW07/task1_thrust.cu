#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/generate.h>
#include <iostream>
#include <hip/hip_runtime.h>

__host__ static __inline__ float rand_01()
{
    return ((float)rand()/RAND_MAX);
}

int main(int argc, char *argv[]) {
    if(argc != 2){
        exit(-1);
    }
    unsigned n = atoi(argv[1]);
    // generate n random float numbers on the host
    thrust::host_vector<float> h_vec(n);
    thrust::generate(h_vec.begin(), h_vec.end(), rand_01);
    
    // transfer data to the device
    thrust::device_vector<float> d_vec(n);
    thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin()); 
    
    //creating cuda timing variables
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    //thrust function call
    float result = thrust::reduce(d_vec.begin(), d_vec.end(), (float)0.0, thrust::plus<float>());
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    
    std::cout << result << std::endl << ms << std::endl;
    return 0;
}

