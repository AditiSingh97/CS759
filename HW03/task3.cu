#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include "vscale.cuh"

int main(int argc, char* argv[]) {
    if(argc != 2){
        exit(-1);
    }

    unsigned int n = atoi(argv[1]);
    //Allocate vectors in host memory
    float *a = new float[n];
    float *b = new float[n];

    // Allocate vectors in device memory
    float *da, *db;
    hipMalloc(&da, n*sizeof(float));
    hipMalloc(&db, n*sizeof(float));

    //random number generation
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dist1(-10.0,10.0);
    std::uniform_real_distribution<> dist2(0.0,1.0);

    for(unsigned int i = 0; i < n; i++){
        a[i] = dist1(gen);
        b[i] = dist2(gen);
    }

    // Copy vectors from host memory to device memory
    hipMemcpy(da, a, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(db, b, n*sizeof(float), hipMemcpyHostToDevice);

    //creating cuda timing variables
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    //kernel call
    vscale<<<n/512, 512>>>(da, db, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    //copy from device memory to host memory
    hipMemcpy(b, db, n*sizeof(float), hipMemcpyDeviceToHost);

    //printing results
    std::printf("%f\n%f\n%f\n", ms, b[0], b[n-1]);

    //freeing memory
    delete [] a;
    delete [] b;
    hipFree(da);
    hipFree(db);

    return 0;
}