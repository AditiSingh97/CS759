#include <cstdio>
#include <random>
#include <hip/hip_runtime.h>

#include "scan.cuh"

int main(int argc, char* argv[]) {
    if(argc != 3){
        exit(-1);
    }

    unsigned int n = atoi(argv[1]);
    unsigned int threads_per_block = atoi(argv[2]);

    //allocating host memory for input matrices
    float *input, *output;
    hipMallocManaged(&input, n*sizeof(float));
    hipMallocManaged(&output, n*sizeof(float));

    //random number generation
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dist(-1, 1);

    for(unsigned int i = 0; i < n; i++){
	    input[i] = dist(gen);
	    output[i] = 0.0;
    }

    //creating cuda timing variables
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    scan(input, output, n, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Get the elapsed time in milliseconds
    float ms = 0.0;
    hipEventElapsedTime(&ms, start, stop);
    printf("%f\n%f\n", output[n-1], ms);

    hipFree(input);
    hipFree(output);
    return 0;
}

