#include <cstdio>
#include <random>
#include <hip/hip_runtime.h>

#include "scan.cuh"

int main(int argc, char* argv[]) {
    if(argc != 3){
        exit(-1);
    }

    unsigned int n = atoi(argv[1]);
    unsigned int threads_per_block = atoi(argv[2]);

    //allocating host memory for input matrices
    float *input, *output;
    hipMallocManaged(&input, n*sizeof(float));
    hipMallocManaged(&output, n*sizeof(float));

    for(unsigned int i = 0; i < n; i++){
	    input[i] = (float)i+1;
	    output[i] = 0.0;
    }

    scan(input, output, n, threads_per_block);

    for(unsigned int i = 0; i < n; i++){
	    printf("output[%u] = %f\n", i, output[i]);
    }

    hipFree(input);
    hipFree(output);
    return 0;
}

