#include <cstdio>
#include <random>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "mmul.h"

int main(int argc, char* argv[]) {
    if(argc != 3){
        exit(-1);
    }

    unsigned int n = atoi(argv[1]);
    unsigned int n_tests = atoi(argv[2]);

    unsigned int SIZE = n*n;
    //allocating host memory for input matrices
    float *A, *B, *C;
    hipMallocManaged(&A, SIZE*sizeof(float));
    hipMallocManaged(&B, SIZE*sizeof(float));
    hipMallocManaged(&C, SIZE*sizeof(float));

    //random number generation
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dist(-1, 1);

    for(unsigned int i = 0; i < SIZE; i++){
        A[i] = dist(gen);
        B[i] = dist(gen);
        C[i] = dist(gen);
    }
    
    //creating cuda timing variables
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float ms = 0.0;
    for(int i = 0; i < n_tests; i++){
        hipEventRecord(start);
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        //kernel call
        mmul(handle, A, B, C, n);
        hipblasDestroy(handle);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
    

        // Get the elapsed time in milliseconds
        float ms1 = 0.0;
        hipEventElapsedTime(&ms1, start, stop);
        ms += ms1;
    }
    
    printf("Average time: %.6f\n", ms/(float)n_tests);

    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}
