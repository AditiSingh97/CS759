#include <cstdio>
#include <random>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "mmul.h"

int main(int argc, char* argv[]) {
    if(argc != 3){
        exit(-1);
    }

    unsigned int n = atoi(argv[1]);
    unsigned int n_tests = atoi(argv[2]);

    unsigned int SIZE = n*n;
    //allocating host memory for input matrices
    float *A, *B, *C;

    //random number generation
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dist(-1, 1);

    //creating cuda timing variables
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float ms = 0.0;
    for(unsigned int i = 0; i < n_tests; i++){
	hipMallocManaged(&A, SIZE*sizeof(float));
        hipMallocManaged(&B, SIZE*sizeof(float));
        hipMallocManaged(&C, SIZE*sizeof(float));
        for(unsigned int j = 0; j < SIZE; j++){
            A[j] = dist(gen);
            B[j] = dist(gen);
            C[j] = dist(gen);
        }
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        hipEventRecord(start);
        //kernel call
        mmul(handle, A, B, C, n);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipblasDestroy(handle);
    

        // Get the elapsed time in milliseconds
        float ms1 = 0.0;
        hipEventElapsedTime(&ms1, start, stop);
        ms += ms1;
        hipFree(A);
        hipFree(B);
        hipFree(C);
    }
    
    printf("Average time: %.6f\n", ms/float(n_tests));
    return 0;
}
