#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "mmul.h"

void mmul(hipblasHandle_t handle, const float* A, const float* B, float* C, int n){
    const float alpha = 1.0;
    const float beta = 1.0;
    const float *ptr1 = &alpha;
    const float *ptr2 = &beta;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, ptr1, A, n, B, n, ptr2, C, n);
    hipDeviceSynchronize();

}
