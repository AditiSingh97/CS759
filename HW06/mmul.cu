#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "mmul.h"

void mmul(hipblasHandle_t handle, const float* A, const float* B, float* C, int n){
    const float alpha = 1.0;
    const float beta = 1.0;
    const float *alpha_ptr = &alpha;
    const float *beta_ptr = &beta;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha_ptr, A, n, B, n, beta_ptr, C, n);
    hipDeviceSynchronize();
}
